#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>

#pragma comment(lib, "cuda.lib")
#pragma comment(lib, "cudart.lib")
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipblas.h>

using namespace std;

#define blocksize 8

/*storing matrix*/
void matrix_read(double *L, int dimension){
	FILE *fp;
	int row, col;

	fp = fopen("randomMatrix_1000.txt", "r");//open output file
	if (fp == NULL)//open failed
		return;

	for (row = 0; row < dimension; row++){
		for (col = 0; col < dimension; col++)
		if (fscanf(fp, "%f,", &L[row * dimension + col]) == EOF) break;//read data

		if (feof(fp)) break;//if the file is over
	}

	fclose(fp);//close file

}

__global__ void nodiag_normalize(double *A, double *I, int n, int i){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
	if (x == i && x!=y){
		I[x*n + y] /= A[i*n + i];
		A[x*n + y] /= A[i*n + i];
	}
	
}

__global__ void diag_normalize(double *A, double *I, int n, int i){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
	if (x == y && x == i){
		I[x*n + y] /= A[i*n + i];
		A[x*n + y] /= A[i*n + i];
	}

}

__global__ void gaussjordan(double *A, double *I, int n, int i)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n){
		if (x != i){
			I[x*n + y] -= I[i*n + y] * A[x*n + i];
			if (y != i){
				A[x*n + y] -= A[i*n + y] * A[x*n + i];
			}	 
		}
	}

}

__global__ void set_zero(double *A, double *I, int n, int i){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n){
		if (x != i){
			if (y == i){
				A[x*n + y] = 0;
			}
		}
	}
}

void savetofile(double *A, string s, int n, int h)
{
	std::ofstream plik;
	plik.open(s);

	for (int j = 0; j<h; j++){
		for (int i = 0; i<h; i++){
			plik << A[j*n + i] << "\t";
		}
		plik << endl;
	}
	plik.close();
}

int main()
{
	const int n = 1000;
	// creating input
	double *iL = new double[n*n];
	double *L = new double[n*n];
	matrix_read(L, n);
	//savetofile(L, "L.txt", n, n);

	cout << "inv\n";
	double *d_A, *d_L, *I, *dI;
	float time;
	hipError_t err;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int ddsize = n*n*sizeof(double);

	dim3 threadsPerBlock(blocksize, blocksize);
	dim3 numBlocks((n + blocksize - 1) / blocksize, (n + blocksize - 1) / blocksize);
	// memory allocation    
	err = hipMalloc((void**)&d_A, ddsize);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
	err = hipMalloc((void**)&dI, ddsize);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
	I = new double[n*n];

	for (int i = 0; i<n; i++){
		for (int j = 0; j<n; j++){
			if (i == j) I[i*n + i] = 1.0;
			else I[i*n + j] = 0.0;
		}
	}

	//copy data from CPU to GPU
	err = hipMemcpy(d_A, L, ddsize, hipMemcpyHostToDevice);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
	err = hipMemcpy(dI, I, ddsize, hipMemcpyHostToDevice);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }

	//timer start
	hipEventRecord(start, 0);

	// L^(-1)    
	for (int i = 0; i<n; i++){
		nodiag_normalize << <numBlocks, threadsPerBlock >> >(d_A, dI, n, i);
		diag_normalize << <numBlocks, threadsPerBlock >> >(d_A, dI, n, i);
		gaussjordan << <numBlocks, threadsPerBlock >> >(d_A, dI, n, i);
		set_zero << <numBlocks, threadsPerBlock >> >(d_A, dI, n, i);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//copy data from GPU to CPU
	err = hipMemcpy(iL, dI, ddsize, hipMemcpyDeviceToHost);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
	err = hipMemcpy(I, d_A, ddsize, hipMemcpyDeviceToHost);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }

	cout << "Cuda Time - inverse: " << time << "ms\n";
	savetofile(iL, "inv.txt", n, n);
	//savetofile(I, "I.txt", n, n);
	hipFree(d_A);
	hipFree(dI);

	double *c = new double[n*n];
	for (int i = 0; i<n; i++)  
	for (int j = 0; j<n; j++)  
	{
		c[i*n+j] = 0;  //put the initial value to zero
		for (int x = 0; x<n; x++)  
			c[i*n + j] = c[i*n + j] + L[i*n+x] * iL[x*n + j];  //matrix multiplication
	}
	savetofile(c, "c.txt", n, n);

	delete[]I;
	delete[]L;
	delete[]iL;

	system("Pause");
	return 0;
}
