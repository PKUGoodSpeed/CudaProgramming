#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define PERR(call) \
  if (call) {\
   fprintf(stderr, "%s:%d Error [%s] on "#call"\n", __FILE__, __LINE__,\
      hipGetErrorString(hipGetLastError()));\
   exit(1);\
  }
#define ERRCHECK \
  if (hipPeekAtLastError()) { \
    fprintf(stderr, "%s:%d Error [%s]\n", __FILE__, __LINE__,\
       hipGetErrorString(hipGetLastError()));\
    exit(1);\
  }

__global__ void
inv_kernel(float *a_i, float *c_o, int n)
{
  int *p = (int *)malloc(3*sizeof(int));
  int *info = (int *)malloc(sizeof(int));
  int batch;
  hipblasHandle_t hdl;
  hipblasStatus_t status = hipblasCreate(&hdl);
  printf("handle %d n = %d\n", status, n);

  info[0] = 0;
  batch = 1;
  float **a = (float **)malloc(sizeof(float *));
  *a = a_i;
  const float **aconst = (const float **)a;
  float **c = (float **)malloc(sizeof(float *));
  *c = c_o;
  status = hipblasSgetrfBatched(hdl, n, a, n, p, info, batch);
  __syncthreads();
  printf("rf %d info %d\n", status, info[0]);
  status = hipblasSgetriBatched(hdl, n, aconst, n, p,
      c, n, info, batch);
  __syncthreads();
  printf("ri %d info %d\n", status, info[0]);
  hipblasDestroy(hdl);
  printf("done\n");
}
static void
run_inv(float *in, float *out, int n)
{
  float *a_d, *c_d;

  PERR(hipMalloc(&a_d, n*n*sizeof(float)));
  PERR(hipMalloc(&c_d, n*n*sizeof(float)));
  PERR(hipMemcpy(a_d, in, n*n*sizeof(float), hipMemcpyHostToDevice));

  inv_kernel<<<1, 1>>>(a_d, c_d, n);

  hipDeviceSynchronize();
  ERRCHECK;

  PERR(hipMemcpy(out, c_d, n*n*sizeof(float), hipMemcpyDeviceToHost));
  PERR(hipFree(a_d));
  PERR(hipFree(c_d));
}

int
main(int argc, char **argv)
{
  float c[9];
  float a[] = {
    1,   2,   3,
    0,   4,   5,
    1,   0,   6 };

  run_inv(a, c, 3);
  for (int i = 0; i < 3; i++){
    for (int j = 0; j < 3; j++) printf("%f, ",c[(3*i)+j]);
    printf("\n");}

  return 0;
}
