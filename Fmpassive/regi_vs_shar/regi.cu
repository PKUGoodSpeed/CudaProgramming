#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
#define gpu_copy(x, y) thrust::copy((x).begin(), (x).end(), (y).begin())
#define gpu_copy_to(x, y, pos) thrust::copy((x).begin(), (x).end(), (y).begin() + (pos))
#define def_dvec(t) thrust::device_vector<t>

using namespace std;

const int VEC_SIZE = 12288;

__global__ void init(){}

__device__ float doThings(int t,int sz, float *input){
    float ans = 0;
    for(int i=0;i<10;++i){
        ans += input[(i+t)%sz];
    }
    return ans;
}

__global__ void process(int N_step, float *input, float *output){
    float local_data[VEC_SIZE];
    thrust::copy(thrust::device, input, input + VEC_SIZE, local_data);
    for(int t=0;t<N_step;++t){
        output[t] = doThings(t, VEC_SIZE, local_data);
    }
}

int main(int argc, char *argv[]){
    srand(0);
    int num_step = 1024*1024;
    if(argc > 1) num_step = stoi(argv[1]);
    /* For measuing the time */
    hipEvent_t start, stop;
    float cuda_time;
    hipEventCreate(&start);   // creating the event 1
    hipEventCreate(&stop);    // creating the event 2
    vector<float> hin(VEC_SIZE), hout(num_step);
    def_dvec(float) din(VEC_SIZE), dout(num_step);
    generate(hin.begin(), hin.end(), [](){return float(rand())/RAND_MAX;});
    init<<<1024,1>>>();
    gpu_copy(hin, din);
    hipEventRecord(start, 0);
    process<<<1024, 1>>>(num_step, to_ptr(din), to_ptr(dout));
    hipEventRecord(stop, 0);                  // Stop time measuring
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop); // Saving the time measured
    cout<<"Time Usage for running the kernel is: "<<cuda_time/1000<<"s"<<endl;
    gpu_copy(dout, hout);
    cout<<"Showing the answer:"<<endl;
    for(int i=0;i<num_step;i+=num_step/10) cout<<hout[i]<<' ';
    cout<<endl;
    return 0;
}