#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>


using namespace std;
const int mod = 1E6;
int main(){
	int N = 1<<25;
	srand(0);
	thrust::host_vector<int> nums(N);
	thrust::device_vector<int> d_vec(N);
	clock_t start_time = clock(), end_time;
	/* Generating data testing*/
	thrust::generate(d_vec.begin(),d_vec.end(), [&](){return rand()%mod;});
	end_time = clock();
	cout<<"=====================Generating Data Time Usage========================"<<endl<<endl;
	cout<<"\t\t"<<double(end_time-start_time)/CLOCKS_PER_SEC<<" s\t\t"<<endl<<endl;
	cout<<"======================================================================="<<endl;
	thrust::copy(d_vec.begin(), d_vec.end(), nums.begin());
	for(int i=0, block = N/10;i<N; i+=block) cout<<nums[i]<<' ';
	cout<<endl<<endl<<endl;
	
	/* Sorting testing*/
	start_time = clock();
	thrust::sort(d_vec.begin(), d_vec.end());
	end_time = clock();
	cout<<"===========================Sorting Time Usage=========================="<<endl<<endl;
	cout<<"\t\t"<<double(end_time-start_time)/CLOCKS_PER_SEC<<" s\t\t"<<endl<<endl;
	thrust::copy(d_vec.begin(), d_vec.end(), nums.begin());
	cout<<"======================================================================="<<endl;
	for(int i=0, block = N/10;i<N; i+=block) cout<<nums[i]<<' ';
	cout<<endl<<endl<<endl;

	return 0;
}
