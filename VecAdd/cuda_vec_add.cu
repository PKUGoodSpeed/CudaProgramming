#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
using namespace std;

__global__ void vec_add(int N, int *A, int *B, int *C){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    assert( i<N );
    C[i] = A[i] + B[i];
}

int main(int argc, char *argv[]){
    if(argc <= 1){
        cout<<"Error: No Input Dimension";
        return 0;
    }
    srand(0);
    int N = stoi(argv[1]), block_size = 256;
    int n_block = (N+block_size-1)/block_size;
    int *A = new int [N], B = new int [N], C = new int [N];
    for(int i=0;i<N;++i) A[i] = rand()%50;
    for(int i=0;i<N;++i) B[i] = rand()%50;
    clock_t start_time,end_time;
    // Record the starting time
    start_time = clock();
    int *dA, *dB, *dC;
    hipMalloc((void **)&dA, N*sizeof(int));
    hipMalloc((void **)&dB, N*sizeof(int));
    hipMalloc((void **)&dC, N*sizeof(int));

    // Copy data to divice
    hipMemcpy(dA, A, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, N*sizeof(int), hipMemcpyHostToDevice);

    // Running code on GPUs
    vec_add<<<n_block, block_size>>>(N, dA, dB, dC);
    hipMemcpy(C, dC, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    // Record the ending time
    end_time = clock();
    double dt = double(end_time - start_time)/CLOCKS_PER_SEC;
    cout<<"Time Usage: "<<dt<<"s\nResults:\n";
    int stride = N/10;
    for(int i=0;i<N;i+=stride) cout<<C[i]<<' ';
    cout<<endl;
    delete [] A;
    delete [] B;
    delete [] C;
    return 0;
}
