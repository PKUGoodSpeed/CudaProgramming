#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
using namespace std;

class TryClass{
public:
    __global__ void vec_add(int N, int *A, int *B, int *C){
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if(i < N) C[0] = A[i] * B[i];
    }
};

int main(int argc, char *argv[]){
    int N = 10, block_size = 16;
    srand(0);
    if(argc > 1) N = stoi(argv[1]);
    int n_block = (N+block_size-1)/block_size;
    int *A = new int [N], *B = new int [N], *C = new int [1];
    for(int i=0;i<N;++i) A[i] = 1;
    for(int i=0;i<N;++i) B[i] = 1;
   
    int *dA, *dB, *dC;
    hipMalloc((void **)&dA, N*sizeof(int));
    hipMalloc((void **)&dB, N*sizeof(int));
    :hipMalloc((void **)&dC, N*sizeof(int));

    // Copy data to divice
    hipMemcpy(dA, A, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, N*sizeof(int), hipMemcpyHostToDevice);
    mid_time1 = clock();

    // Running code on GPUs
    TryClass().vec_add<<<n_block, block_size>>>(N, dA, dB, dC);
    mid_time2 = clock();
    hipMemcpy(C, dC, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    // Record the ending time
    end_time = clock();
    double dt = double(end_time - start_time)/CLOCKS_PER_SEC;
    double dt_trans = double(mid_time1 + end_time - start_time - mid_time2)/CLOCKS_PER_SEC;
    cout<<"Data Transfer Time Usage: "<<dt_trans<<"s"<<endl;
    cout<<"Total Time Usage: "<<dt<<"s\nResults:\n";
    int stride = N/10;
    for(int i=0;i<N;i+=stride) cout<<C[i]<<' ';
    cout<<endl;
    delete [] A;
    delete [] B;
    delete [] C;
    return 0;
}
